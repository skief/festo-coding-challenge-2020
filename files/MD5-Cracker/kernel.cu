
#include "hip/hip_runtime.h"
#include ""
#include "md5.h"
#include "timer.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>

#define CUDA_CHECK_ERROR                                                       \
    do {                                                                       \
        const hipError_t err = hipGetLastError();                            \
        if (err != hipSuccess) {                                              \
            const char *const err_str = hipGetErrorString(err);               \
            std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 1   \
                      << ": " << err_str << " (" << err << ")" << std::endl;   \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while(0)

#define N_THREADS 32 * 32
#define N_GRID 1000

int pos(char s) {
    for (int i = 0; i < posLen; i++) {
        if (possible[i] == s) {
            return i;
        }
    }
    return 0;
}


int main() {
    dim3 threads(N_THREADS); // (32 * 32);
    dim3 grid(N_GRID); //(1000);

    unsigned char* h_out = (unsigned char*)malloc(sizeof(unsigned char) * 8);
    unsigned char* h_input = (unsigned char*)malloc(sizeof(unsigned char) * 8 * N_THREADS * N_GRID); //[] = "sQyWhn3wsQyWhn4x";
    unsigned char* h_hit = (unsigned char*)malloc(sizeof(unsigned char) * N_THREADS * N_GRID);
    //unsigned char h_pattern[8] = { 0x00, 0x2a, 0x8a, 0x8b, 0x23, 0xd0, 0x3e, 0x70 };
    //unsigned char h_pattern[8] = { 0x00, 0x2a, 0x8a, 0x8b, 0x23, 0xd0, 0x3e, 0x70 };
    unsigned char h_pattern[8] = { 0xa8, 0x4b, 0xa6, 0x51, 0xfd, 0x12, 0x2e, 0xf5 };
    unsigned char* d_out;
    unsigned char* d_input;
    unsigned char* d_hit;
    unsigned char* d_pattern;
    hipMalloc(&d_out, sizeof(unsigned char) * 8);
    hipMalloc(&d_input, sizeof(unsigned char) * N_GRID * N_THREADS * 8);
    hipMalloc(&d_hit, sizeof(unsigned char) * N_GRID * N_THREADS);
    hipMalloc(&d_pattern, sizeof(unsigned char) * 16);

    hipMemcpy(d_pattern, h_pattern, sizeof(unsigned char) * 8, hipMemcpyHostToDevice);

    md5_init();

    int count = 0;
    int first = 0;

    timer_tp start = timer_now();
    timer_tp stop;
    timer_tp progstart = timer_now();

    float allpos = posLen * posLen;
    int posTested = 0;
    unsigned int hashesDone = 0;
	
    for (int a = 0; a < posLen; a++) {
        for (int b = 0; b < posLen; b++) {
            stop = timer_now();
            float timePerPos = timer_elapsed(start, stop);
            posTested += 1;
            float possibilitiesLeft = allpos - posTested;
            start = timer_now();
            
            hashesDone = 0;

            for (int c = 0; c < posLen; c++) {
                for (int d = 0; d < posLen; d++) {
                    for (int e = 0; e < posLen; e++) {
                        for (int f = 0; f < posLen; f++) {
                            for (int g = 0; g < posLen; g++) {
                                h_input[8 * count + 0] = possible[a];
                                h_input[8 * count + 1] = possible[b];
                                h_input[8 * count + 2] = possible[c];
                                h_input[8 * count + 3] = possible[d];
                                h_input[8 * count + 4] = possible[e];
                                h_input[8 * count + 5] = possible[f];
                                h_input[8 * count + 6] = possible[g];

                                count += 1;
                                if (count == N_GRID * N_THREADS) {
                                    hipMemcpy(d_input, h_input, sizeof(unsigned char) * N_GRID * N_THREADS * 8, hipMemcpyHostToDevice);

                                    if (first != 0) {
                                        hipMemcpy(h_hit, d_hit, sizeof(unsigned char) * N_THREADS * N_GRID, hipMemcpyDeviceToHost);

                                        for (int i = 0; i < N_THREADS * N_GRID; i++) {
                                           // std::cout << (int)h_hit[i] << " ";
                                            if (h_hit[i] == 1) {
                                                hipMemcpy(h_out, d_out, sizeof(unsigned char) * 8, hipMemcpyDeviceToHost);
                                                for (int j = 0; j < 8; j++) {
                                                    printf("%c", h_out[j]);
                                                }
                                                std::cout << std::endl;
                                                return;
                                            }
                                        }
                                    }

                                    Parallel_Hash << <grid, threads >> > (d_input, d_out, d_hit, d_pattern, 8);
                                    first = 1;
                                    hashesDone += 1;
                                    count = 0;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
    CUDA_CHECK_ERROR;
	return 0;
}